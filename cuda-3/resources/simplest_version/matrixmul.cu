#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "dev_array.h"
#include <math.h>
#include <helper_timer.h>

using namespace std;

int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    int N = 16;
    int SIZE = N*N;

    // Allocate memory on the host
    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);
    vector<float> h_C(SIZE);

    // Initialize matrices on the host
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = sin(i);
            h_B[i*N+j] = cos(j);
        }
    }

    // Allocate memory on the device
    dev_array<float> d_A(SIZE);
    dev_array<float> d_B(SIZE);
    dev_array<float> d_C(SIZE);

    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);
    
    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
  
    sdkStartTimer(&timer);

    matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();

    // Czas kopiowania danych do pamięci hosta wliczam w czas prowadzenia obliczen
    // bo jezeli nie bedziemy miec tych danych, to nic nam z tych obliczen.
    sdkStopTimer(&timer);
    float gpu_time = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);

    float *cpu_C;
    cpu_C=new float[SIZE];

    sdkStartTimer(&timer);
    // Now do the matrix multiplication on the CPU
    float sum;
    for (int row=0; row<N; row++){
        for (int col=0; col<N; col++){
            sum = 0.f;
            for (int n=0; n<N; n++){
                sum += h_A[row*N+n]*h_B[n*N+col];
            }
            cpu_C[row*N+col] = sum;
        }
    }

    sdkStopTimer(&timer);
    float cpu_time = sdkGetTimerValue(&timer);

    double err = 0;
    // Check the result and make sure it is correct
    for (int ROW=0; ROW < N; ROW++){
        for (int COL=0; COL < N; COL++){
            err += cpu_C[ROW * N + COL] - h_C[ROW * N + COL];
        }
    }

    cout << "Error: " << err << endl;
    cout << "GPU: " << gpu_time << "  CPU: " << cpu_time << '\n';


    sdkDeleteTimer(&timer);
    return 0;
}
