
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

// #include "gputimer.h"
struct GpuTimer {
  hipEvent_t start;
  hipEvent_t stop;

  GpuTimer() {
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }

  ~GpuTimer() {
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  void Start() { hipEventRecord(start, 0); }

  void Stop() { hipEventRecord(stop, 0); }

  float Elapsed() {
    float elapsed;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
  }
};

#define N 1048576 // 512, 1024, ... 16384, 32768, 65536, 131072

void host_add(int *a, int *b, int *c, int size) {
  for (int idx = 0; idx < size; idx++)
    c[idx] = a[idx] + b[idx];
}

__global__ void device_add(int *a, int *b, int *c) {

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

// basically just fills the array with index.
void fill_array(int *data, int size) {
  for (int i = 0; i < size; ++i) {
    data[i] = i;
  }
}

void print_output(int *a, int *b, int *c, int size) {
  for (int idx = 0; idx < size; idx++)
    printf("\n %d + %d  = %d", a[idx], b[idx], c[idx]);
}
int main(void) {
  std::cout << "Starting the run...\n";

  std::vector<int> arr_sizes;
  std::vector<int> thread_counts{32, 64, 128, 256, 512};

  int base = 512;
  while (base <= N) {
    arr_sizes.push_back(base);
    base *= 2;
  }

  printf("arrsize,nblocks,nthreads,time\n");

  for (int i = arr_sizes.size() - 1; i >= 0; --i) {
    for (int i_thread = 0; i_thread < thread_counts.size(); ++i_thread) {
      int *a, *b, *c;
      int *d_a, *d_b, *d_c; // device copies of a, b, c
      int threads_per_block = 0, no_of_blocks = 0;
      GpuTimer timer;
      
      int n_elems = arr_sizes[i];
      int size = n_elems * sizeof(int);

      // Alloc space for host copies of a, b, c and setup input values
      a = (int *)malloc(size);
      fill_array(a, n_elems);
      b = (int *)malloc(size);
      fill_array(b, n_elems);
      c = (int *)malloc(size);

      // Alloc space for device copies of a, b, c
      hipMalloc((void **)&d_a, size);
      hipMalloc((void **)&d_b, size);
      hipMalloc((void **)&d_c, size);

      // Copy inputs to device
      hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
      hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

      threads_per_block = thread_counts[i_thread];
      no_of_blocks = arr_sizes[i] / threads_per_block;
      timer.Start();
      device_add<<<no_of_blocks, threads_per_block>>>(d_a, d_b, d_c);
      timer.Stop();

      // Copy result back to host
      hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

      // print_output(a,b,c);
      printf("%d,%d,%d,%f\n", n_elems, no_of_blocks, threads_per_block, timer.Elapsed());
      // printf("N = %d; no_of_blocks = %d; Elapsed time = %f ms\n", N,
      // no_of_blocks,
      //        timer.Elapsed());

      free(a);
      free(b);
      free(c);
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
    }
  }

  return 0;
}
