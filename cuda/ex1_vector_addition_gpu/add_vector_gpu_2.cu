
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

// #include "gputimer.h"
struct GpuTimer {
  hipEvent_t start;
  hipEvent_t stop;

  GpuTimer() {
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }

  ~GpuTimer() {
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  void Start() { hipEventRecord(start, 0); }

  void Stop() { hipEventRecord(stop, 0); }

  float Elapsed() {
    float elapsed;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
  }
};

#define N 1048576 // 512, 1024, ... 16384, 32768, 65536, 131072

void host_add(int *a, int *b, int *c, int size) {
  for (int idx = 0; idx < size; idx++)
    c[idx] = a[idx] + b[idx];
}

__global__ void device_add(int *a, int *b, int *c) {

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

// basically just fills the array with index.
void fill_array(int *data, int size) {
  for (int idx = 0; idx < size; idx++)
    data[idx] = idx;
}

void print_output(int *a, int *b, int *c, int size) {
  for (int idx = 0; idx < size; idx++)
    printf("\n %d + %d  = %d", a[idx], b[idx], c[idx]);
}
int main(void) {
  std::cout << "Starting the run...\n";

  std::vector<int> arr_sizes;

  int base = 512;
  while (base <= N) {
    arr_sizes.push_back(base);
    base *= 2;
  }

  printf("arrsize,nblocks,time\n");

  for (int i = arr_sizes.size() - 1; i >= 0; --i) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int threads_per_block = 0, no_of_blocks = 0;
    GpuTimer timer;

    int size = arr_sizes[i] * sizeof(int);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size);
    fill_array(a, size);
    b = (int *)malloc(size);
    fill_array(b, size);
    c = (int *)malloc(size);

    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    threads_per_block = 512;
    no_of_blocks = arr_sizes[i] / threads_per_block;
    timer.Start();
    device_add<<<no_of_blocks, threads_per_block>>>(d_a, d_b, d_c);
    timer.Stop();

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // print_output(a,b,c);
    printf("%d,%d,%f\n", arr_sizes[i], no_of_blocks, timer.Elapsed());
    // printf("N = %d; no_of_blocks = %d; Elapsed time = %f ms\n", N,
    // no_of_blocks,
    //        timer.Elapsed());

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
  }

  return 0;
}
